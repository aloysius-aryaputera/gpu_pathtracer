#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <string>
#include <time.h>

#include <thrust/sort.h>

#include "external/libjpeg_cpp/jpeg.h"

#include "model/bvh/bvh_build.h"
#include "model/camera.h"
#include "model/data_structure/local_vector.h"
#include "model/geometry/sphere.h"
#include "model/geometry/triangle.h"
#include "model/grid/bounding_box.h"
#include "model/grid/cell.h"
#include "model/grid/grid.h"
#include "model/material.h"
#include "model/ray.h"
#include "model/scene.h"
#include "model/vector_and_matrix/vec3.h"
#include "render/pathtracing.h"
#include "util/image_util.h"
#include "util/read_file_util.h"
#include "util/read_image_util.h"
#include "util/string_util.h"
#include "world_lib.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void create_scene(
  Scene** scene, Camera** camera, Grid** grid, int *num_objects
) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    *(scene) = new Scene(camera[0], grid[0], num_objects[0]);
  }
}

__global__ void render_init(
  int im_width, int im_height, hiprandState *rand_state
) {
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  if ((j >= im_width) || (i >= im_height)) {
    return;
  }
  int pixel_index = i * im_width + j;
  //Each thread gets same seed, a different sequence number, no offset
  hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void free_world(
  Scene** scene, Grid **grid, Primitive **geom_array, Camera **camera, int n
) {
    for (int i = 0; i < n; i++){
      delete *(geom_array + i);
    }
    delete *camera;
    delete *grid;
    delete *scene;
}

__global__ void test(Primitive **object_array, int n) {
  printf("%u\n", object_array[0] -> get_bounding_box() -> morton_code);
  printf("%d\n", object_array[0] -> get_bounding_box() -> morton_code < object_array[1] -> get_bounding_box() -> morton_code);
  printf("%u\n", object_array[1] -> get_bounding_box() -> morton_code);
  printf("%d\n", object_array[1] -> get_bounding_box() -> morton_code < object_array[2] -> get_bounding_box() -> morton_code);
  printf("%u\n", object_array[2] -> get_bounding_box() -> morton_code);
  printf("%u\n", object_array[n / 2] -> get_bounding_box() -> morton_code);
  printf("%u\n", object_array[n - 5] -> get_bounding_box() -> morton_code);
  printf("%d\n", object_array[n - 5] -> get_bounding_box() -> morton_code < object_array[n - 3] -> get_bounding_box() -> morton_code);
  printf("%u\n", object_array[n - 3] -> get_bounding_box() -> morton_code);
  printf("%d\n", object_array[n - 3] -> get_bounding_box() -> morton_code < object_array[n - 1] -> get_bounding_box() -> morton_code);
  printf("%u\n", object_array[n - 1] -> get_bounding_box() -> morton_code);
}

__global__ void my_clz(unsigned int x) {
  int result = __clz(x);
  printf("clz = %d\n", result);
  printf("clz_2 = %d\n", __clz(41));
  printf("******************************************\n");
}

int main(int argc, char **argv) {
  hipDeviceSetLimit(hipLimitMallocHeapSize, 1024ULL*1024ULL*1024ULL*4ULL);

  std::string process;
  time_t my_time = time(NULL);
  clock_t first_start, start;

  process = "Rendering project";
  print_start_process(process, first_start);

  std::string input_folder_path = argv[1];
  std::string obj_filename = argv[2];
  std::string texture_bg_path = argv[3];
  std::string image_output_path = argv[4];

  int im_width = std::stoi(argv[5]);
  int im_height = std::stoi(argv[6]);
  int pathtracing_sample_size = std::stoi(argv[7]);
  int pathtracing_level = std::stoi(argv[8]);
  float eye_x = std::stof(argv[9]);
  float eye_y = std::stof(argv[10]);
  float eye_z = std::stof(argv[11]);
  float center_x = std::stof(argv[12]);
  float center_y = std::stof(argv[13]);
  float center_z = std::stof(argv[14]);
  float up_x = std::stof(argv[15]);
  float up_y = std::stof(argv[16]);
  float up_z = std::stof(argv[17]);
  float fovy = std::stof(argv[18]);
  float aperture = std::stof(argv[19]);
  float focus_dist = std::stof(argv[20]);

  float sky_emission_r = std::stof(argv[21]);
  float sky_emission_g = std::stof(argv[22]);
  float sky_emission_b = std::stof(argv[23]);

  int *n_cell_x, *n_cell_y, *n_cell_z;
  int max_n_cell_x = 120, max_n_cell_y = 120, max_n_cell_z = 120;
  int tx = 8, ty = 8, tx2 = 8, ty2 = 8, tz2 = 8, max_num_objects_per_cell = 1000;

  Scene** my_scene;
  Grid** my_grid;
  Cell** my_cell;
  Primitive **my_geom, **my_cell_geom;
  unsigned int *morton_code_list;
  Material **my_material;
  Camera **my_camera;
  vec3 *image_output;

  int num_pixels = im_width * im_height;
  int max_num_materials = 100;
  int num_vertices, num_faces, num_vt, num_vn;
  size_t image_size = num_pixels * sizeof(vec3);
  hiprandState *rand_state;
  size_t rand_state_size = num_pixels * sizeof(hiprandState);

  float *ka_x, *ka_y, *ka_z, *kd_x, *kd_y, *kd_z;
  float *ks_x, *ks_y, *ks_z, *ke_x, *ke_y, *ke_z, *n_s, *n_i, *t_r;
  float *tf_x, *tf_y, *tf_z;
  float *material_image_r, *material_image_g, *material_image_b;
  int *num_materials;
  int *material_image_height_diffuse, *material_image_width_diffuse, \
    *material_image_offset_diffuse;
  int *material_image_height_specular, *material_image_width_specular, \
    *material_image_offset_specular;
  int *material_image_height_n_s, *material_image_width_n_s, \
    *material_image_offset_n_s;

  float *bg_texture_r, *bg_texture_g, *bg_texture_b;
  int bg_height, bg_width;

  // int a1 = std::stoi(argv[24]);
  // int a2 = std::stoi(argv[25]);
  // printf("a1 = %d\n", a1);
  // printf("a2 = %d\n", a2);
  // my_clz<<<1, 1>>>((unsigned int)(a1) ^ (unsigned int)(a2));
  // checkCudaErrors(hipGetLastError());
  // checkCudaErrors(hipDeviceSynchronize());

  // printf("================================================================\n");

  ///////////////////////////////////////////////////////////////////////////
  // For offline testing
  ///////////////////////////////////////////////////////////////////////////
  // float ka_x[100], ka_y[100], ka_z[100], kd_x[100], kd_y[100], kd_z[100];
  // float ks_x[100], ks_y[100], ks_z[100], ke_x[100], ke_y[100], ke_z[100];
  // float material_image_r[1000], material_image_g[1000], material_image_b[1000];
  // int num_materials[1], material_image_height[100], material_image_width[100], material_image_offset[100];
  // int len_texture[1];
  ///////////////////////////////////////////////////////////////////////////

  start = clock();
  process = "Extracting background texture";
  print_start_process(process, start);
  extract_single_image_requirement(
    input_folder_path, texture_bg_path, bg_height, bg_width
  );

  checkCudaErrors(hipMallocManaged(
    (void **)&bg_texture_r, bg_height * bg_width * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&bg_texture_g, bg_height * bg_width * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&bg_texture_b, bg_height * bg_width * sizeof(float)));

  int next_idx = 0;
  extract_single_image(
    input_folder_path, texture_bg_path, bg_texture_r, bg_texture_g,
    bg_texture_b, next_idx
  );
  print_end_process(process, start);

  std::vector <std::string> material_file_name_array, material_name;

  checkCudaErrors(hipMallocManaged((void **)&num_materials, sizeof(int)));

  checkCudaErrors(hipMallocManaged((void **)&ka_x, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&ka_y, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&ka_z, max_num_materials * sizeof(float)));

  checkCudaErrors(hipMallocManaged((void **)&kd_x, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&kd_y, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&kd_z, max_num_materials * sizeof(float)));

  checkCudaErrors(hipMallocManaged((void **)&ks_x, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&ks_y, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&ks_z, max_num_materials * sizeof(float)));

  checkCudaErrors(hipMallocManaged((void **)&ke_x, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&ke_y, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&ke_z, max_num_materials * sizeof(float)));

  checkCudaErrors(hipMallocManaged((void **)&tf_x, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&tf_y, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&tf_z, max_num_materials * sizeof(float)));

  checkCudaErrors(hipMallocManaged((void **)&t_r, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&n_s, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&n_i, max_num_materials * sizeof(float)));

  start = clock();
  process = "Extracting material file names";
  print_start_process(process, start);
  extract_material_file_names(
    input_folder_path,
    obj_filename,
    material_file_name_array
  );
  print_end_process(process, start);

  std::vector <std::string> texture_file_name_array;
  std::vector <int> texture_offset_array, texture_height_array, \
    texture_width_array;
  long int texture_length = 0;

  start = clock();
  process = "Extracting texture resource requirements";
  print_start_process(process, start);
  extract_image_resource_requirement(
    input_folder_path,
    material_file_name_array,
    texture_file_name_array,
    texture_offset_array,
    texture_height_array,
    texture_width_array,
    texture_length
  );
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged((void **)&material_image_r, texture_length * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&material_image_g, texture_length * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&material_image_b, texture_length * sizeof(float)));

  start = clock();
  process = "Extracting textures";
  print_start_process(process, start);
  extract_textures(
    input_folder_path,
    texture_file_name_array,
    material_image_r,
    material_image_g,
    material_image_b
  );
  print_end_process(process, start);

  start = clock();
  process = "Extracting the number of the elements";
  print_start_process(process, start);
  extract_num_elements(
    input_folder_path, obj_filename,
    num_vertices, num_vt, num_vn, num_faces
  );
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_height_diffuse, max_num_materials * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_width_diffuse, max_num_materials * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_offset_diffuse, max_num_materials * sizeof(int)));

  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_height_specular,
    max_num_materials * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_width_specular,
    max_num_materials * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_offset_specular,
    max_num_materials * sizeof(int)));

  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_height_n_s,
    max_num_materials * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_width_n_s,
    max_num_materials * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_offset_n_s,
    max_num_materials * sizeof(int)));

  start = clock();
  process = "Extracting material data";
  print_start_process(process, start);
  extract_material_data(
    input_folder_path,
    material_file_name_array,
    texture_file_name_array,
    texture_offset_array,
    texture_height_array,
    texture_width_array,
    ka_x, ka_y, ka_z,
    kd_x, kd_y, kd_z,
    ks_x, ks_y, ks_z,
    ke_x, ke_y, ke_z,
    tf_x, tf_y, tf_z,
    t_r, n_s, n_i,
    material_image_height_diffuse, material_image_width_diffuse,
    material_image_offset_diffuse,
    material_image_height_specular, material_image_width_specular,
    material_image_offset_specular,
    material_image_height_n_s, material_image_width_n_s,
    material_image_offset_n_s,
    num_materials,
    material_name
  );
  print_end_process(process, start);

  float *x, *y, *z, *x_norm, *y_norm, *z_norm, *x_tex, *y_tex;
  int *point_1_idx, *point_2_idx, *point_3_idx, \
    *norm_1_idx, *norm_2_idx, *norm_3_idx, \
    *tex_1_idx, *tex_2_idx, *tex_3_idx;
  int *num_triangles, *material_idx;

  checkCudaErrors(hipMallocManaged((void **)&num_triangles, sizeof(int)));

  checkCudaErrors(hipMallocManaged((void **)&material_idx, num_faces * sizeof(int)));

  checkCudaErrors(hipMallocManaged((void **)&x, max(1, num_vertices) * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&y, max(1, num_vertices) * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&z, max(1, num_vertices) * sizeof(float)));

  checkCudaErrors(hipMallocManaged((void **)&x_norm, max(1, num_vn) * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&y_norm, max(1, num_vn) * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&z_norm, max(1, num_vn) * sizeof(float)));

  checkCudaErrors(hipMallocManaged((void **)&x_tex, max(1, num_vt) * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&y_tex, max(1, num_vt) * sizeof(float)));

  checkCudaErrors(hipMallocManaged((void **)&point_1_idx, num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&point_2_idx, num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&point_3_idx, num_faces * sizeof(int)));

  checkCudaErrors(hipMallocManaged((void **)&norm_1_idx, num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&norm_2_idx, num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&norm_3_idx, num_faces * sizeof(int)));

  checkCudaErrors(hipMallocManaged((void **)&tex_1_idx, num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&tex_2_idx, num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&tex_3_idx, num_faces * sizeof(int)));

  start = clock();
  process = "Reading OBJ file";
  print_start_process(process, start);
  extract_triangle_data(
    input_folder_path,
    obj_filename,
    x, y, z,
    x_norm, y_norm, z_norm,
    x_tex, y_tex,
    point_1_idx, point_2_idx, point_3_idx,
    norm_1_idx, norm_2_idx, norm_3_idx,
    tex_1_idx, tex_2_idx, tex_3_idx,
    material_name,
    material_idx,
    num_triangles,
    num_materials
  );
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged((void **)&my_camera, sizeof(Camera *)));

  start = clock();
  process = "Creating the camera";
  print_start_process(process, start);
  create_camera<<<1, 1>>>(
    my_camera,
    eye_x, eye_y, eye_z,
    center_x, center_y, center_z,
    up_x, up_y, up_z, fovy,
    im_width, im_height,
    aperture, focus_dist
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged(
    (void **)&my_material, max_num_materials * sizeof(Material *)));

  start = clock();
  process = "Creating the materials";
  print_start_process(process, start);
  create_material<<<1, num_materials[0]>>>(
    my_material,
    ka_x, ka_y, ka_z,
    kd_x, kd_y, kd_z,
    ks_x, ks_y, ks_z,
    ke_x, ke_y, ke_z,
    tf_x, tf_y, tf_z,
    t_r, n_s, n_i,
    material_image_height_diffuse,
    material_image_width_diffuse,
    material_image_offset_diffuse,
    material_image_height_specular,
    material_image_width_specular,
    material_image_offset_specular,
    material_image_height_n_s,
    material_image_width_n_s,
    material_image_offset_n_s,
    material_image_r,
    material_image_g,
    material_image_b,
    num_materials
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipFree(ka_x));
  checkCudaErrors(hipFree(ka_y));
  checkCudaErrors(hipFree(ka_z));
  checkCudaErrors(hipFree(kd_x));
  checkCudaErrors(hipFree(kd_y));
  checkCudaErrors(hipFree(kd_z));
  checkCudaErrors(hipFree(ks_x));
  checkCudaErrors(hipFree(ks_y));
  checkCudaErrors(hipFree(ks_z));
  checkCudaErrors(hipFree(ke_x));
  checkCudaErrors(hipFree(ke_y));
  checkCudaErrors(hipFree(ke_z));
  checkCudaErrors(hipFree(tf_x));
  checkCudaErrors(hipFree(tf_y));
  checkCudaErrors(hipFree(tf_z));
  checkCudaErrors(hipFree(n_s));
  checkCudaErrors(hipFree(material_image_height_diffuse));
  checkCudaErrors(hipFree(material_image_width_diffuse));
  checkCudaErrors(hipFree(material_image_offset_diffuse));
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMallocManaged(
    (void **)&my_geom, num_triangles[0] * sizeof(Primitive *)));
  checkCudaErrors(hipMallocManaged(
    (void **)&morton_code_list, num_triangles[0] * sizeof(unsigned int)));

  start = clock();
  process = "Creating the world";
  print_start_process(process, start);
  dim3 blocks_world(num_triangles[0] / 256 + 1);
  dim3 threads_world(256);
  create_world<<<blocks_world, threads_world>>>(
    my_geom, my_material,
    x, y, z,
    x_norm, y_norm, z_norm,
    x_tex, y_tex,
    point_1_idx, point_2_idx, point_3_idx,
    norm_1_idx, norm_2_idx, norm_3_idx,
    tex_1_idx, tex_2_idx, tex_3_idx,
    material_idx,
    num_triangles
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipFree(x));
  checkCudaErrors(hipFree(y));
  checkCudaErrors(hipFree(z));
  checkCudaErrors(hipFree(x_norm));
  checkCudaErrors(hipFree(y_norm));
  checkCudaErrors(hipFree(z_norm));
  checkCudaErrors(hipFree(point_1_idx));
  checkCudaErrors(hipFree(point_2_idx));
  checkCudaErrors(hipFree(point_3_idx));
  checkCudaErrors(hipFree(norm_1_idx));
  checkCudaErrors(hipFree(norm_2_idx));
  checkCudaErrors(hipFree(norm_3_idx));
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMallocManaged((void **)&my_grid, sizeof(Grid *)));
  checkCudaErrors(hipMallocManaged((void **)&n_cell_x, sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&n_cell_y, sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&n_cell_z, sizeof(int)));

  start = clock();
  process = "Preparing the grid";
  print_start_process(process, start);
  prepare_grid<<<1, 1>>>(
    my_camera, my_geom, num_triangles,
    n_cell_x, n_cell_y, n_cell_z,
    max_n_cell_x, max_n_cell_y, max_n_cell_z
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(
    hipMallocManaged(
      (void **)&my_cell,
      n_cell_x[0] * n_cell_y[0] * n_cell_z[0] * sizeof(Cell*)));

  start = clock();
  process = "Creating the grid";
  print_start_process(process, start);
  create_grid<<<1, 1>>>(
    my_camera, my_grid, my_geom, num_triangles, my_cell, n_cell_x, n_cell_y,
    n_cell_z, max_n_cell_x, max_n_cell_y, max_n_cell_z, max_num_objects_per_cell
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Computing the morton code of every bounding box";
  print_start_process(process, start);
  compute_morton_code_batch<<<blocks_world, threads_world>>>(
    my_geom, my_grid, num_triangles[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  auto ff = []  __device__ (Primitive* obj_1, Primitive* obj_2) {
    return obj_1 -> get_bounding_box() -> morton_code < \
      obj_2 -> get_bounding_box() -> morton_code;
  };

  start = clock();
  process = "Sorting the objects based on morton code";
  print_start_process(process, start);
  thrust::stable_sort(thrust::device, my_geom, my_geom + num_triangles[0], ff);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  Node** node_list;
  Leaf** leaf_list;
  checkCudaErrors(hipMallocManaged(
    (void **)&node_list, (num_triangles[0] - 1) * sizeof(Node *)));
  checkCudaErrors(hipMallocManaged(
    (void **)&leaf_list, num_triangles[0] * sizeof(Leaf *)));

  start = clock();
  process = "Building leaves";
  print_start_process(process, start);
  build_leaf_list<<<blocks_world, threads_world>>>(
    leaf_list, my_geom, num_triangles[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Building nodes";
  print_start_process(process, start);
  build_node_list<<<blocks_world, threads_world>>>(
    node_list, num_triangles[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Extracting morton codes";
  print_start_process(process, start);
  extract_morton_code_list<<<blocks_world, threads_world>>>(
    my_geom, morton_code_list, num_triangles[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  // start = clock();
  // process = "Set node relationship";
  // print_start_process(process, start);
  // set_node_relationship<<<blocks_world, threads_world>>>(
  //   node_list, leaf_list, morton_code_list, num_triangles[0]
  // );
  // checkCudaErrors(hipGetLastError());
  // checkCudaErrors(hipDeviceSynchronize());
  // print_end_process(process, start);
  //
  // start = clock();
  // process = "Compute node bounding boxes";
  // print_start_process(process, start);
  // compute_node_bounding_boxes<<<blocks_world, threads_world>>>(
  //   leaf_list, node_list, num_triangles[0]
  // );
  // checkCudaErrors(hipGetLastError());
  // checkCudaErrors(hipDeviceSynchronize());
  // print_end_process(process, start);

  size_t cell_geom_size = max_num_objects_per_cell * \
    n_cell_x[0] * n_cell_y[0] * n_cell_z[0] * sizeof(Primitive*);
  checkCudaErrors(hipMallocManaged((void **)&my_cell_geom, cell_geom_size));
  dim3 blocks2(n_cell_x[0] / tx2 + 1, n_cell_y[0] / ty2 + 1, n_cell_z[0] / tz2 + 1);
  dim3 threads2(tx2, ty2, tz2);

  start = clock();
  process = "Building cell array";
  print_start_process(process, start);
  build_cell_array<<<blocks2, threads2>>>(my_grid, my_cell_geom);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Inserting objects into the grid";
  print_start_process(process, start);
  insert_objects<<<blocks2, threads2>>>(my_grid);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged((void **)&my_scene, sizeof(Scene *)));

  start = clock();
  process = "Creating the scene";
  print_start_process(process, start);
  create_scene<<<1, 1>>>(my_scene, my_camera, my_grid, num_triangles);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  dim3 blocks(im_width / tx + 1, im_height / ty + 1);
  dim3 threads(tx, ty);
  checkCudaErrors(hipMallocManaged((void **)&rand_state, rand_state_size));

  start = clock();
  process = "Preparing the rendering process";
  print_start_process(process, start);
  render_init<<<blocks, threads>>>(im_width, im_height, rand_state);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  vec3 sky_emission = vec3(sky_emission_r, sky_emission_g, sky_emission_b);
  checkCudaErrors(hipMallocManaged((void **)&image_output, image_size));

  start = clock();
  process = "Rendering";
  print_start_process(process, start);
  render<<<blocks, threads>>>(
    image_output, my_scene, rand_state, pathtracing_sample_size,
    pathtracing_level, sky_emission, bg_height, bg_width,
    bg_texture_r, bg_texture_g, bg_texture_b
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Saving image";
  print_start_process(process, start);
  save_image(image_output, im_width, im_height, image_output_path);
  print_end_process(process, start);
  checkCudaErrors(hipDeviceSynchronize());

  start = clock();
  process = "Cleaning";
  print_start_process(process, start);
  // free_world<<<1,1>>>(my_scene, my_grid, my_geom, my_camera, max_num_faces);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipFree(my_scene));
  checkCudaErrors(hipFree(my_grid));
  checkCudaErrors(hipFree(my_camera));
  checkCudaErrors(hipFree(my_geom));
  checkCudaErrors(hipFree(my_material));
  checkCudaErrors(hipFree(num_triangles));
  checkCudaErrors(hipFree(n_cell_x));
  checkCudaErrors(hipFree(n_cell_y));
  checkCudaErrors(hipFree(n_cell_z));
  checkCudaErrors(hipFree(rand_state));
  checkCudaErrors(hipFree(material_image_r));
  checkCudaErrors(hipFree(material_image_g));
  checkCudaErrors(hipFree(material_image_b));
  checkCudaErrors(hipFree(bg_texture_r));
  checkCudaErrors(hipFree(bg_texture_g));
  checkCudaErrors(hipFree(bg_texture_b));
  checkCudaErrors(hipFree(image_output));
  print_end_process(process, start);

  print_end_process("Rendering project", first_start);

  hipDeviceReset();

  return 0;
}
