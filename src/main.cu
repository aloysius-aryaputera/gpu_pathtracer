#include <iostream>
#include <math.h>

#include "model/vector_and_matrix/vec3.h"
#include "render/pathtracing.h"
#include "util/image_util.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

int main(int argc, char **argv) {
  int nx = 640, ny = 640, tx = 8, ty = 8;
  int num_pixels = nx*ny;
  size_t fb_size = 3 * num_pixels * sizeof(float);

  // allocate FB
  float *fb;
  checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

  dim3 blocks(nx / tx + 1, ny / ty + 1);
  dim3 threads(tx, ty);
  render<<<blocks, threads>>>(fb, nx, ny);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  save_image(fb, nx, ny, argv[1]);

  checkCudaErrors(hipFree(fb));

  return 0;
}
