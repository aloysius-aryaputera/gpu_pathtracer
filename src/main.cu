#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <string>
#include <time.h>

#include "model/camera.h"
#include "model/data_structure/local_vector.h"
#include "model/geometry/sphere.h"
#include "model/geometry/triangle.h"
#include "model/grid/bounding_box.h"
#include "model/grid/cell.h"
#include "model/grid/grid.h"
#include "model/material.h"
#include "model/ray.h"
#include "model/scene.h"
#include "model/vector_and_matrix/vec3.h"
#include "render/pathtracing.h"
#include "util/image_util.h"
#include "util/read_file_util.h"
#include "world_lib.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void create_scene(
  Scene** scene, Camera** camera, Grid** grid, int *num_objects
) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    *(scene) = new Scene(camera[0], grid[0], num_objects[0]);
  }
}

__global__ void render_init(
  int im_width, int im_height, hiprandState *rand_state, int *progress
) {
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  if (i == 0 && j == 0) {
    progress[0] = 0;
  }
  if ((j >= im_width) || (i >= im_height)) {
    return;
  }
  int pixel_index = i * im_width + j;
  //Each thread gets same seed, a different sequence number, no offset
  hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void free_world(
  Scene** scene, Grid **grid, Primitive **geom_array, Camera **camera, int n
) {
    for (int i = 0; i < n; i++){
      delete *(geom_array + i);
    }
    delete *camera;
    delete *grid;
    delete *scene;
}

int main(int argc, char **argv) {
  time_t my_time = time(NULL);
  printf("Started at %s\n\n", ctime(&my_time));

  int im_width = std::stoi(argv[3]), im_height = std::stoi(argv[4]);
  int tx = std::stoi(argv[5]), ty = std::stoi(argv[6]);
  int *n_cell_x, *n_cell_y, *n_cell_z;
  int max_n_cell_x = 60, max_n_cell_y = 60, max_n_cell_z = 60;
  int tx2 = 8, ty2 = 8, max_num_objects_per_cell = 500, *progress;

  printf("image width = %d, image height = %d\n\n", im_width, im_height);

  BoundingBox** my_cell_bounding_box;
  Scene** my_scene;
  Grid** my_grid;
  Cell** my_cell;
  Primitive **my_geom, **my_cell_geom;
  Camera **my_camera;
  vec3 *fb;
  int num_pixels = im_width * im_height;
  int max_num_vertices = 60000, max_num_faces = 110000;
  size_t fb_size = num_pixels * sizeof(vec3);
  hiprandState *rand_state;
  size_t rand_state_size = num_pixels * sizeof(hiprandState);
  size_t cell_geom_size = max_num_objects_per_cell * (max_n_cell_x) * \
    (max_n_cell_y) * (max_n_cell_z) * sizeof(Primitive*);
  size_t cell_bounding_box_size = (max_n_cell_x) * (max_n_cell_y) * \
    (max_n_cell_z) * sizeof(BoundingBox*);
  clock_t start, stop;

  start = clock();
  float *x, *y, *z, *x_norm, *y_norm, *z_norm;
  int *point_1_idx, *point_2_idx, *point_3_idx, \
    *norm_1_idx, *norm_2_idx, *norm_3_idx;
  int *num_triangles;

  // float x[100000], y[100000], z[100000];
  // int point_1_idx[100000], point_2_idx[100000], point_3_idx[100000];
  // int num_triangles[1];

  checkCudaErrors(hipMallocManaged((void **)&num_triangles, sizeof(int)));

  checkCudaErrors(hipMallocManaged((void **)&x, max_num_vertices * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&y, max_num_vertices * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&z, max_num_vertices * sizeof(float)));

  checkCudaErrors(hipMallocManaged((void **)&x_norm, max_num_vertices * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&y_norm, max_num_vertices * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&z_norm, max_num_vertices * sizeof(float)));

  checkCudaErrors(hipMallocManaged((void **)&point_1_idx, max_num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&point_2_idx, max_num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&point_3_idx, max_num_faces * sizeof(int)));

  checkCudaErrors(hipMallocManaged((void **)&norm_1_idx, max_num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&norm_2_idx, max_num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&norm_3_idx, max_num_faces * sizeof(int)));

  printf("Reading OBJ file!\n");
  extract_triangle_data_2(
    argv[2], x, y, z, x_norm, y_norm, z_norm,
    point_1_idx, point_2_idx, point_3_idx, norm_1_idx, norm_2_idx, norm_3_idx,
    num_triangles
  );
  my_time = time(NULL);
  printf("OBJ file read at %s!\n\n", ctime(&my_time));

  checkCudaErrors(hipMallocManaged((void **)&my_geom, max_num_faces * sizeof(Primitive *)));
  checkCudaErrors(hipMallocManaged((void **)&my_camera, sizeof(Camera *)));

  printf("Creating the world!\n");
  create_world_3<<<1, 1>>>(
    my_camera, my_geom,
    x, y, z,
    x_norm, y_norm, z_norm,
    point_1_idx, point_2_idx, point_3_idx,
    norm_1_idx, norm_2_idx, norm_3_idx,
    num_triangles, im_width, im_height
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  my_time = time(NULL);
  printf("World created at %s!\n\n", ctime(&my_time));

  checkCudaErrors(hipFree(x));
  checkCudaErrors(hipFree(y));
  checkCudaErrors(hipFree(z));
  checkCudaErrors(hipFree(x_norm));
  checkCudaErrors(hipFree(y_norm));
  checkCudaErrors(hipFree(z_norm));
  checkCudaErrors(hipFree(point_1_idx));
  checkCudaErrors(hipFree(point_2_idx));
  checkCudaErrors(hipFree(point_3_idx));
  checkCudaErrors(hipFree(norm_1_idx));
  checkCudaErrors(hipFree(norm_2_idx));
  checkCudaErrors(hipFree(norm_3_idx));
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMallocManaged((void **)&my_grid, sizeof(Grid *)));
  checkCudaErrors(hipMallocManaged((void **)&my_cell, max_n_cell_x * max_n_cell_y * max_n_cell_z * sizeof(Cell *)));
  checkCudaErrors(hipMallocManaged((void **)&n_cell_x, sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&n_cell_y, sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&n_cell_z, sizeof(int)));

  printf("Creating the grid!\n");
  create_grid<<<1, 1>>>(
    my_camera, my_grid, my_geom, num_triangles, my_cell, n_cell_x, n_cell_y,
    n_cell_z, max_n_cell_x, max_n_cell_y, max_n_cell_z, max_num_objects_per_cell
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  my_time = time(NULL);
  printf("Grid created at %s!\n\n", ctime(&my_time));

  dim3 blocks2(n_cell_x[0] / tx2 + 1, n_cell_y[0] / ty2 + 1);
  dim3 threads2(tx2, ty2);
  checkCudaErrors(hipMallocManaged((void **)&my_cell_geom, cell_geom_size));
  checkCudaErrors(hipMallocManaged(
    (void **)&my_cell_bounding_box, cell_bounding_box_size));

  printf("Building cell array!\n");
  build_cell_array<<<blocks2, threads2>>>(my_grid, my_cell_geom, my_cell_bounding_box);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  my_time = time(NULL);
  printf("Cell array built at %s!\n\n", ctime(&my_time));

  printf("Inserting objects into the grid!\n");
  insert_objects<<<blocks2, threads2>>>(my_grid);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  my_time = time(NULL);
  printf("Objects inserted into the grid at %s!\n\n", ctime(&my_time));

  checkCudaErrors(hipMallocManaged((void **)&my_scene, sizeof(Scene *)));

  printf("Creating scene!\n");
  create_scene<<<1, 1>>>(my_scene, my_camera, my_grid, num_triangles);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  my_time = time(NULL);
  printf("Scene created at %s!\n\n", ctime(&my_time));

  dim3 blocks(im_width / tx + 1, im_height / ty + 1);
  dim3 threads(tx, ty);
  checkCudaErrors(hipMallocManaged((void **)&rand_state, rand_state_size));
  checkCudaErrors(hipMallocManaged((void **)&progress, sizeof(int)));

  printf("Preparing the rendering process!\n");
  render_init<<<blocks, threads>>>(im_width, im_height, rand_state, progress);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  my_time = time(NULL);
  printf("Rendering process is ready to start at %s!\n\n", ctime(&my_time));

  vec3 sky_emission = vec3(1, 1, 1);
  checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

  printf("Rendering started!\n");
  render<<<blocks, threads>>>(
    fb, my_scene, rand_state, std::stoi(argv[7]), std::stoi(argv[8]),
    sky_emission, progress
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  my_time = time(NULL);
  printf("Rendering done at %s!\n\n", ctime(&my_time));

  printf("Saving image!\n");
  save_image(fb, im_width, im_height, argv[1]);
  my_time = time(NULL);
  printf("Image saved at %s!\n\n", ctime(&my_time));

  stop = clock();
  double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
  printf("\nThe rendering took %5.5f seconds.\n", timer_seconds);

  checkCudaErrors(hipDeviceSynchronize());
  printf("Do cleaning!\n");
  free_world<<<1,1>>>(my_scene, my_grid, my_geom, my_camera, max_num_faces);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipFree(my_scene));
  checkCudaErrors(hipFree(my_grid));
  checkCudaErrors(hipFree(my_camera));
  checkCudaErrors(hipFree(my_geom));
  checkCudaErrors(hipFree(num_triangles));
  checkCudaErrors(hipFree(n_cell_x));
  checkCudaErrors(hipFree(n_cell_y));
  checkCudaErrors(hipFree(n_cell_z));
  checkCudaErrors(hipFree(rand_state));
  checkCudaErrors(hipFree(fb));
  my_time = time(NULL);
  printf("Cleaning done at %s!\n\n", ctime(&my_time));

  hipDeviceReset();

  return 0;
}
