#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <string>
#include <time.h>

#include <thrust/sort.h>

#include "external/libjpeg_cpp/jpeg.h"

#include "input/input_param.h"
#include "input/read_file_util.h"
#include "input/read_image_util.h"
#include "lib/world.h"
#include "model/bvh/bvh.h"
#include "model/bvh/bvh_building.h"
#include "model/bvh/bvh_building_photon.h"
#include "model/bvh/bvh_building_pts.h"
#include "model/bvh/bvh_traversal_photon.h"
#include "model/camera.h"
#include "model/geometry/primitive.h"
#include "model/geometry/triangle.h"
#include "model/geometry/triangle_operations.h"
#include "model/grid/bounding_box.h"
#include "model/grid/bounding_box_operations.h"
#include "model/material/material.h"
#include "model/object/object.h"
#include "model/object/object_operations.h"
#include "model/point/point.h"
#include "model/point/point_operations.h"
#include "model/point/ppm_hit_point.h"
#include "model/point/sss_point.h"
#include "model/ray/ray.h"
#include "model/vector_and_matrix/mat3.h"
#include "model/vector_and_matrix/vec3.h"
#include "render/pathtracing.h"
#include "render/pathtracing_target_geom_operations.h"
#include "render/ppm/image_output.h"
#include "render/ppm/photon_pass.h"
#include "render/ppm/ray_tracing_pass.h"
#include "render/transparent_geom_operations.h"
#include "util/general.h"
#include "util/image_util.h"
#include "util/string_util.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(
  hipError_t result, char const *const func, const char *const file, 
	int const line
) {
  if (result) {
    std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
    file << ":" << line << " '" << func << "' \n";
    // Make sure we call CUDA Device Reset before exiting
    hipDeviceReset();
    exit(99);
  }
}

int main(int argc, char **argv) {
  hipDeviceSetLimit(hipLimitMallocHeapSize, 1024ULL*1024ULL*1024ULL*4ULL);

  std::string process;
  time_t my_time = time(NULL);
  clock_t first_start, start;

  process = "Rendering project";
  print_start_process(process, first_start);

  std::string master_file_path = argv[1];
  std::string image_output_path = argv[2];

  InputParam input_param = InputParam();
  input_param.extract_parameters(master_file_path);

  std::string input_folder_path = input_param.input_folder_path;
  std::string obj_filename = input_param.obj_filename;
  std::string texture_bg_path = input_param.texture_bg_path;

  int im_width = input_param.image_width;
  int im_height = input_param.image_height;

  int render_mode = input_param.render_mode;

  int ppm_num_photon_per_pass = input_param.ppm_num_photon_per_pass;
  int ppm_num_pass = input_param.ppm_num_pass;
  int ppm_max_bounce = input_param.ppm_max_bounce;
  float ppm_alpha = input_param.ppm_alpha;
  float ppm_radius_scaling_factor = input_param.ppm_radius_scaling_factor;
  int ppm_image_output_iteration = input_param.ppm_image_output_iteration;

  int pathtracing_sample_size = input_param.pathtracing_sample_size;
  int pathtracing_level = input_param.pathtracing_level;
  int dof_sample_size = input_param.dof_sample_size;
  float eye_x = input_param.eye_x;
  float eye_y = input_param.eye_y;
  float eye_z = input_param.eye_z;
  float center_x = input_param.center_x;
  float center_y = input_param.center_y;
  float center_z = input_param.center_z;
  float up_x = input_param.up_x;
  float up_y = input_param.up_y;
  float up_z = input_param.up_z;
  float fovy = input_param.fovy;
  float aperture = input_param.aperture;
  float focus_dist = input_param.focus_dist;

  float sky_emission_r = input_param.sky_emission_r;
  float sky_emission_g = input_param.sky_emission_g;
  float sky_emission_b = input_param.sky_emission_b;

  int sss_pts_per_object = input_param.sss_pts_per_object;
  float hittable_pdf_weight = input_param.hittable_pdf_weight;

  int tx = 8, ty = 8;

  BoundingBox **world_bounding_box, **target_world_bounding_box;
  BoundingBox **transparent_world_bounding_box;
  Primitive **my_geom;
  Primitive **target_geom_list, **transparent_geom_list;
  Object **my_objects;
  unsigned int *morton_code_list;
  Material **my_material;
  Camera **my_camera;
  Point **sss_pts;
  vec3 *image_output;

  int num_pixels = im_width * im_height;
  int max_num_materials = 100;
  int num_objects, num_vertices, num_faces, num_vt, num_vn;
  int *num_sss_objects, *num_target_geom, *num_transparent_geom;
  size_t image_size = num_pixels * sizeof(vec3);
  hiprandState *rand_state_sss, *rand_state_image;

  bool *sss_object_marker_array;
  int *pt_offset_array, *num_pt_array;

  float *ka_x, *ka_y, *ka_z, *kd_x, *kd_y, *kd_z;
  float *ks_x, *ks_y, *ks_z, *ke_x, *ke_y, *ke_z, *n_s, *n_i, *t_r;
  float *tf_x, *tf_y, *tf_z;
  float *path_length;
  float *material_image_r, *material_image_g, *material_image_b;
  float *bm;
  float *scattering_coef, *absorption_coef, *g;
  int *num_materials;
  int *material_image_height_diffuse, *material_image_width_diffuse, \
    *material_image_offset_diffuse, *material_priority;
  int *material_image_height_specular, *material_image_width_specular, \
    *material_image_offset_specular;
  int *material_image_height_n_s, *material_image_width_n_s, \
    *material_image_offset_n_s;
  int *material_image_height_emission, *material_image_width_emission, \
    *material_image_offset_emission;
  int *material_image_height_bump, *material_image_width_bump, \
    *material_image_offset_bump;
  vec3 *tangent, *bitangent;

  float *bg_texture_r, *bg_texture_g, *bg_texture_b;
  int bg_height, bg_width;

  start = clock();
  process = "Extracting background texture";
  print_start_process(process, start);
  extract_single_image_requirement(
    input_folder_path, texture_bg_path, bg_height, bg_width
  );

  checkCudaErrors(hipMallocManaged(
    (void **)&bg_texture_r, bg_height * bg_width * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&bg_texture_g, bg_height * bg_width * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&bg_texture_b, bg_height * bg_width * sizeof(float)));

  int next_idx = 0;
  extract_single_image(
    input_folder_path, texture_bg_path, bg_texture_r, bg_texture_g,
    bg_texture_b, next_idx
  );
  print_end_process(process, start);

  std::vector <std::string> material_file_name_array, material_name;

  checkCudaErrors(hipMallocManaged((void **)&num_materials, sizeof(int)));

  start = clock();
  process = "Extracting material file names";
  print_start_process(process, start);
  extract_material_file_names(
    input_folder_path,
    obj_filename,
    material_file_name_array
  );
  print_end_process(process, start);

  std::vector <std::string> texture_file_name_array;
  std::vector <int> texture_offset_array, texture_height_array, \
    texture_width_array;
  long int texture_length = 0;

  start = clock();
  process = "Extracting texture resource requirements";
  print_start_process(process, start);
  extract_image_resource_requirement(
    input_folder_path,
    material_file_name_array,
    texture_file_name_array,
    texture_offset_array,
    texture_height_array,
    texture_width_array,
    texture_length
  );
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_r, texture_length * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_g, texture_length * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_b, texture_length * sizeof(float)));

  start = clock();
  process = "Extracting textures";
  print_start_process(process, start);
  extract_textures(
    input_folder_path,
    texture_file_name_array,
    material_image_r,
    material_image_g,
    material_image_b
  );
  print_end_process(process, start);

  start = clock();
  process = "Extracting the number of the elements";
  print_start_process(process, start);
  extract_num_elements(
    input_folder_path, obj_filename,
    num_objects, num_vertices, num_vt, num_vn, num_faces
  );
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged(
    (void **)&ka_x, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&ka_y, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&ka_z, max_num_materials * sizeof(float)));

  checkCudaErrors(hipMallocManaged(
    (void **)&kd_x, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&kd_y, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&kd_z, max_num_materials * sizeof(float)));

  checkCudaErrors(hipMallocManaged(
    (void **)&ks_x, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&ks_y, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&ks_z, max_num_materials * sizeof(float)));

  checkCudaErrors(hipMallocManaged(
    (void **)&ke_x, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&ke_y, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&ke_z, max_num_materials * sizeof(float)));

  checkCudaErrors(hipMallocManaged(
    (void **)&tf_x, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&tf_y, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&tf_z, max_num_materials * sizeof(float)));

  checkCudaErrors(hipMallocManaged(
    (void **)&path_length, max_num_materials * sizeof(float)));

  checkCudaErrors(hipMallocManaged(
    (void **)&bm, max_num_materials * sizeof(float)));

  checkCudaErrors(hipMallocManaged(
    (void **)&t_r, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&n_s, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&n_i, max_num_materials * sizeof(float)));

  checkCudaErrors(hipMallocManaged(
    (void **)&scattering_coef, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&absorption_coef, max_num_materials * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&g, max_num_materials * sizeof(float)));

  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_height_diffuse,
    max_num_materials * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_width_diffuse,
    max_num_materials * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_offset_diffuse,
    max_num_materials * sizeof(int)));

  checkCudaErrors(hipMallocManaged(
    (void **)&material_priority, max_num_materials * sizeof(int)));

  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_height_specular,
    max_num_materials * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_width_specular,
    max_num_materials * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_offset_specular,
    max_num_materials * sizeof(int)));

  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_height_n_s,
    max_num_materials * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_width_n_s,
    max_num_materials * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_offset_n_s,
    max_num_materials * sizeof(int)));

  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_height_emission,
    max_num_materials * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_width_emission,
    max_num_materials * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_offset_emission,
    max_num_materials * sizeof(int)));

  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_height_bump,
    max_num_materials * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_width_bump,
    max_num_materials * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_image_offset_bump,
    max_num_materials * sizeof(int)));

  start = clock();
  process = "Extracting material data";
  print_start_process(process, start);
  extract_material_data(
    input_folder_path,
    material_file_name_array,
    texture_file_name_array,
    texture_offset_array,
    texture_height_array,
    texture_width_array,
    ka_x, ka_y, ka_z,
    kd_x, kd_y, kd_z,
    ks_x, ks_y, ks_z,
    ke_x, ke_y, ke_z,
    tf_x, tf_y, tf_z,
    path_length,
    t_r, n_s, n_i, bm,
    scattering_coef, absorption_coef, g,
    material_priority,
    material_image_height_diffuse, material_image_width_diffuse,
    material_image_offset_diffuse,
    material_image_height_specular, material_image_width_specular,
    material_image_offset_specular,
    material_image_height_emission, material_image_width_emission,
    material_image_offset_emission,
    material_image_height_n_s, material_image_width_n_s,
    material_image_offset_n_s,
    material_image_height_bump, material_image_width_bump,
    material_image_offset_bump,
    num_materials,
    material_name
  );
  print_end_process(process, start);

  float *x, *y, *z, *x_norm, *y_norm, *z_norm, *x_tex, *y_tex;
  int *point_1_idx, *point_2_idx, *point_3_idx, \
    *norm_1_idx, *norm_2_idx, *norm_3_idx, \
    *tex_1_idx, *tex_2_idx, *tex_3_idx;
  int *num_triangles, *material_idx;
  int *object_num_primitives, *object_primitive_offset_idx;
  int *triangle_object_idx;
  float *triangle_area, *accumulated_triangle_area;

  checkCudaErrors(hipMallocManaged(
    (void **)&num_triangles, sizeof(int)));

  checkCudaErrors(hipMallocManaged(
    (void **)&object_num_primitives, num_objects * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&object_primitive_offset_idx, num_objects * sizeof(int)));

  checkCudaErrors(hipMallocManaged(
    (void **)&triangle_object_idx, num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&material_idx, num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&triangle_area, num_faces * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&accumulated_triangle_area, num_faces * sizeof(float)));

  checkCudaErrors(hipMallocManaged(
    (void **)&x, max(1, num_vertices) * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&y, max(1, num_vertices) * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&z, max(1, num_vertices) * sizeof(float)));

  checkCudaErrors(hipMallocManaged(
    (void **)&x_norm, max(1, num_vn) * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&y_norm, max(1, num_vn) * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&z_norm, max(1, num_vn) * sizeof(float)));

  checkCudaErrors(hipMallocManaged(
    (void **)&x_tex, max(1, num_vt) * sizeof(float)));
  checkCudaErrors(hipMallocManaged(
    (void **)&y_tex, max(1, num_vt) * sizeof(float)));

  checkCudaErrors(hipMallocManaged(
    (void **)&point_1_idx, num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&point_2_idx, num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&point_3_idx, num_faces * sizeof(int)));

  checkCudaErrors(hipMallocManaged(
    (void **)&norm_1_idx, num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&norm_2_idx, num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&norm_3_idx, num_faces * sizeof(int)));

  checkCudaErrors(hipMallocManaged(
    (void **)&tex_1_idx, num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&tex_2_idx, num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged(
    (void **)&tex_3_idx, num_faces * sizeof(int)));

  start = clock();
  process = "Reading OBJ file";
  print_start_process(process, start);
  extract_triangle_data(
    input_folder_path,
    obj_filename,
    x, y, z,
    x_norm, y_norm, z_norm,
    x_tex, y_tex,
    point_1_idx, point_2_idx, point_3_idx,
    norm_1_idx, norm_2_idx, norm_3_idx,
    tex_1_idx, tex_2_idx, tex_3_idx,
    material_name,
    material_idx,
    num_triangles,
    num_materials,
    triangle_object_idx,
    object_num_primitives,
    object_primitive_offset_idx
  );
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged((void **)&my_camera, sizeof(Camera *)));

  start = clock();
  process = "Creating the camera";
  print_start_process(process, start);
  create_camera<<<1, 1>>>(
    my_camera,
    eye_x, eye_y, eye_z,
    center_x, center_y, center_z,
    up_x, up_y, up_z, fovy,
    im_width, im_height,
    aperture, focus_dist
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged(
    (void **)&my_material, max_num_materials * sizeof(Material *)));

  start = clock();
  process = "Creating the materials";
  print_start_process(process, start);
  create_material<<<1, num_materials[0]>>>(
    my_material,
    ka_x, ka_y, ka_z,
    kd_x, kd_y, kd_z,
    ks_x, ks_y, ks_z,
    ke_x, ke_y, ke_z,
    tf_x, tf_y, tf_z,
    path_length,
    t_r, n_s, n_i, bm,
    scattering_coef, absorption_coef, g,
    material_priority,
    material_image_height_diffuse,
    material_image_width_diffuse,
    material_image_offset_diffuse,
    material_image_height_specular,
    material_image_width_specular,
    material_image_offset_specular,
    material_image_height_emission,
    material_image_width_emission,
    material_image_offset_emission,
    material_image_height_n_s,
    material_image_width_n_s,
    material_image_offset_n_s,
    material_image_height_bump,
    material_image_width_bump,
    material_image_offset_bump,
    material_image_r,
    material_image_g,
    material_image_b,
    num_materials
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipFree(ka_x));
  checkCudaErrors(hipFree(ka_y));
  checkCudaErrors(hipFree(ka_z));
  checkCudaErrors(hipFree(kd_x));
  checkCudaErrors(hipFree(kd_y));
  checkCudaErrors(hipFree(kd_z));
  checkCudaErrors(hipFree(ks_x));
  checkCudaErrors(hipFree(ks_y));
  checkCudaErrors(hipFree(ks_z));
  checkCudaErrors(hipFree(ke_x));
  checkCudaErrors(hipFree(ke_y));
  checkCudaErrors(hipFree(ke_z));
  checkCudaErrors(hipFree(tf_x));
  checkCudaErrors(hipFree(tf_y));
  checkCudaErrors(hipFree(tf_z));
  checkCudaErrors(hipFree(n_s));
  checkCudaErrors(hipFree(material_image_height_diffuse));
  checkCudaErrors(hipFree(material_image_width_diffuse));
  checkCudaErrors(hipFree(material_image_offset_diffuse));
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMallocManaged(
    (void **)&my_objects, num_objects * sizeof(Object *)));
  checkCudaErrors(hipMallocManaged(
    (void **)&my_geom, num_triangles[0] * sizeof(Primitive *)));
  checkCudaErrors(hipMallocManaged(
    (void **)&morton_code_list, num_triangles[0] * sizeof(unsigned int)));

  start = clock();
  process = "Creating the objects";
  print_start_process(process, start);
  dim3 blocks_object(num_objects + 1);
  dim3 threads_object(1);
  create_objects<<<blocks_object, threads_object>>>(
    my_objects, object_num_primitives, object_primitive_offset_idx,
    triangle_area, accumulated_triangle_area, num_objects
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged(
    (void **)&sss_object_marker_array, num_objects * sizeof(bool)));

  start = clock();
  process = "Creating the world";
  print_start_process(process, start);
  dim3 blocks_world(num_triangles[0] / 1 + 1);
  dim3 threads_world(1);
  create_world<<<blocks_world, threads_world>>>(
    my_geom,
    triangle_area,
    my_objects, triangle_object_idx,
    my_material,
    x, y, z,
    x_norm, y_norm, z_norm,
    x_tex, y_tex,
    point_1_idx, point_2_idx, point_3_idx,
    norm_1_idx, norm_2_idx, norm_3_idx,
    tex_1_idx, tex_2_idx, tex_3_idx,
    material_idx,
    num_triangles,
    sss_object_marker_array,
    sss_pts_per_object
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged(
    (void **)&tangent, num_vertices * sizeof(vec3)));
  checkCudaErrors(hipMallocManaged(
    (void **)&bitangent, num_vertices * sizeof(vec3)));

  start = clock();
  process = "Summing up tangents and bitangents";
  print_start_process(process, start);
  sum_up_tangent_and_bitangent<<<1, 1>>>(
    tangent, bitangent, my_geom, num_triangles[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Assigning tangents";
  print_start_process(process, start);
  assign_tangent<<<num_triangles[0], 1>>>(
    tangent, my_geom, num_triangles[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipFree(x));
  checkCudaErrors(hipFree(y));
  checkCudaErrors(hipFree(z));
  checkCudaErrors(hipFree(x_norm));
  checkCudaErrors(hipFree(y_norm));
  checkCudaErrors(hipFree(z_norm));
  checkCudaErrors(hipFree(point_1_idx));
  checkCudaErrors(hipFree(point_2_idx));
  checkCudaErrors(hipFree(point_3_idx));
  checkCudaErrors(hipFree(norm_1_idx));
  checkCudaErrors(hipFree(norm_2_idx));
  checkCudaErrors(hipFree(norm_3_idx));
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMallocManaged((void **)&num_sss_objects, sizeof(int)));
  checkCudaErrors(
    hipMallocManaged((void **)&pt_offset_array, num_objects * sizeof(int)));
  checkCudaErrors(
    hipMallocManaged((void **)&num_pt_array, num_objects * sizeof(int)));

  start = clock();
  process = "Computing the number of SSS objects";
  print_start_process(process, start);
  compute_num_sss_objects<<<1, 1>>>(
    num_sss_objects, my_objects, pt_offset_array, num_pt_array,
    num_objects, sss_pts_per_object
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  int num_sss_points = sss_pts_per_object * num_sss_objects[0];

  checkCudaErrors(hipMallocManaged
    ((void **)&sss_pts, max(1, num_sss_points) * sizeof(Point*)));

  start = clock();
  process = "Allocating " + std::to_string(num_sss_points) + \
    " points for SSS objects";
  print_start_process(process, start);
  allocate_pts_sss<<<num_objects, 1>>>(
    my_objects, sss_pts, pt_offset_array, num_objects);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged(
    (void **)&world_bounding_box, sizeof(BoundingBox *)));

  checkCudaErrors(hipMallocManaged(
    (void **)&rand_state_sss, max(1, num_sss_points) * sizeof(hiprandState)));

  start = clock();
  process = "Generating hiprand state for SSS points sampling";
  print_start_process(process, start);
  init_curand_state<<<max(1, num_sss_points), 1>>>(
    num_sss_points, rand_state_sss);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  process = "Creating SSS points samplings";
  start = clock();
  print_start_process(process, start);
  for (int i = 0; i < num_objects; i++) {
    create_sss_pts<<<sss_pts_per_object, 1>>>(
      my_objects, my_geom, sss_pts, pt_offset_array, rand_state_sss,
      i, sss_pts_per_object
    );
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
  }
  print_end_process(process, start);

  start = clock();
  process = "Computing the object boundaries";
  print_start_process(process, start);
  compute_object_boundaries_batch<<<num_objects, 1>>>(
    my_objects, num_objects
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Computing the morton code of every point bounding box";
  print_start_process(process, start);
  compute_pts_morton_code_batch<<<blocks_world, threads_world>>>(
    my_objects, sss_pts, num_sss_points
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  auto sort_points = []  __device__ (Point* pt_1, Point* pt_2) {
    return pt_1 -> bounding_box -> morton_code < \
      pt_2 -> bounding_box -> morton_code;
  };

  process = "Sorting the points based on morton code";
  start = clock();
  print_start_process(process, start);
  for (int i = 0; i < num_objects; i++) {
    thrust::stable_sort(
      thrust::device, sss_pts + pt_offset_array[i],
      sss_pts + pt_offset_array[i] + num_pt_array[i],
      sort_points);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
  }
  print_end_process(process, start);

  start = clock();
  process = "Computing sss points offset list";
  print_start_process(process, start);
  compute_sss_pts_offset<<<1, 1>>>(my_objects, num_objects);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  Node** sss_pts_node_list, **sss_pts_leaf_list;
  checkCudaErrors(hipMallocManaged(
    (void **)&sss_pts_node_list,
    max(1, (num_sss_points - num_sss_objects[0])) * sizeof(Node *)));
  checkCudaErrors(hipMallocManaged(
    (void **)&sss_pts_leaf_list,
    max(1, num_sss_points) * sizeof(Node *)));

  process = "Building sss points leaves";
  start = clock();
  print_start_process(process, start);
  for (int i = 0; i < num_objects; i++) {
    build_sss_pts_leaf_list<<<max(1, num_sss_points), 1>>>(
      sss_pts_leaf_list, sss_pts, my_objects, i, pt_offset_array
    );
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
  }
  print_end_process(process, start);

  checkCudaErrors(hipFree(pt_offset_array));
  checkCudaErrors(hipFree(num_pt_array));
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  process = "Building sss points nodes";
  start = clock();
  print_start_process(process, start);
  for (int i = 0; i < num_objects; i++) {
    build_sss_pts_node_list<<<max(1, num_sss_points), 1>>>(
      sss_pts_node_list, my_objects, i
    );
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
  }
  print_end_process(process, start);

  unsigned int *sss_morton_code_list;
  checkCudaErrors(hipMallocManaged(
    (void **)&sss_morton_code_list,
    max(1, num_sss_points) * sizeof(unsigned int)));

  start = clock();
  process = "Extracting the morton codes of the SSS points";
  print_start_process(process, start);
  extract_sss_morton_code_list<<<max(1, num_sss_points), 1>>>(
    sss_pts, sss_morton_code_list, num_sss_points
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  process = "Setting the sss nodes relationship";
  start = clock();
  print_start_process(process, start);
  for (int i = 0; i < num_objects; i++) {
    set_pts_sss_node_relationship<<<max(1, num_sss_points), 1>>>(
      sss_pts_node_list, sss_pts_leaf_list, sss_morton_code_list, my_objects, i
    );
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
  }
  print_end_process(process, start);

  checkCudaErrors(hipFree(sss_morton_code_list));
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  start = clock();
  process = "Compute pts node bounding boxes";
  print_start_process(process, start);
  compute_node_bounding_boxes<<<max(1, num_sss_points), 1>>>(
    sss_pts_leaf_list, sss_pts_node_list, num_sss_points
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Computing the world bounding box";
  print_start_process(process, start);
  compute_world_bounding_box<<<1, 1>>>(
    world_bounding_box, my_geom, num_triangles[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Computing the morton code of every geometry bounding box";
  print_start_process(process, start);
  compute_morton_code_batch<<<blocks_world, threads_world>>>(
    my_geom, world_bounding_box, num_triangles[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  auto sort_geom = []  __device__ (Primitive* obj_1, Primitive* obj_2) {
    return obj_1 -> get_bounding_box() -> morton_code < \
      obj_2 -> get_bounding_box() -> morton_code;
  };

  start = clock();
  process = "Sorting the objects based on morton code";
  print_start_process(process, start);
  thrust::stable_sort(
    thrust::device, my_geom, my_geom + num_triangles[0], sort_geom);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  Node** node_list, **leaf_list;
  checkCudaErrors(hipMallocManaged(
    (void **)&node_list, (num_triangles[0] - 1) * sizeof(Node *)));
  checkCudaErrors(hipMallocManaged(
    (void **)&leaf_list, num_triangles[0] * sizeof(Node *)));

  start = clock();
  process = "Building leaves";
  print_start_process(process, start);
  build_leaf_list<<<blocks_world, threads_world>>>(
    leaf_list, my_geom, num_triangles[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Building nodes";
  print_start_process(process, start);
  build_node_list<<<blocks_world, threads_world>>>(
    node_list, num_triangles[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Extracting morton codes";
  print_start_process(process, start);
  extract_morton_code_list<<<blocks_world, threads_world>>>(
    my_geom, morton_code_list, num_triangles[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Setting node relationship";
  print_start_process(process, start);
  set_node_relationship<<<blocks_world, threads_world>>>(
    node_list, leaf_list, morton_code_list, num_triangles[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipFree(morton_code_list));
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  start = clock();
  process = "Compute node bounding boxes";
  print_start_process(process, start);
  compute_node_bounding_boxes<<<blocks_world, threads_world>>>(
    leaf_list, node_list, num_triangles[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Check";
  print_start_process(process, start);
  check<<<blocks_world, threads_world>>>(
    leaf_list,  node_list,  num_triangles[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  vec3 sky_emission = vec3(sky_emission_r, sky_emission_g, sky_emission_b);

  checkCudaErrors(hipMallocManaged(
    (void **)&num_transparent_geom, sizeof(int)));

  start = clock();
  process = "Computing the number of transparent geometries";
  print_start_process(process, start);
  compute_num_transparent_geom<<<1, 1>>>(
    my_geom, num_triangles[0], num_transparent_geom
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged(
    (void **)&transparent_geom_list, 
    max(1, num_transparent_geom[0]) * sizeof(Primitive *)));

  start = clock();
  process = "Collecting transparent geometries";
  print_start_process(process, start);
  collect_transparent_geom<<<1, 1>>>(
    my_geom, num_triangles[0], transparent_geom_list
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged(
    (void **)&transparent_world_bounding_box, sizeof(BoundingBox *)));

  start = clock();
  process = "Computing the transparent world bounding box";
  print_start_process(process, start);
  compute_world_bounding_box<<<1, 1>>>(
    transparent_world_bounding_box, transparent_geom_list, 
    num_transparent_geom[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Computing the morton code of every transparent geometry bounding box";
  print_start_process(process, start);
  compute_morton_code_batch<<<blocks_world, threads_world>>>(
    transparent_geom_list, transparent_world_bounding_box, 
    num_transparent_geom[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Sorting the transparent geometries based on morton code";
  print_start_process(process, start);
  thrust::stable_sort(
    thrust::device, transparent_geom_list, 
    transparent_geom_list + num_transparent_geom[0],
    sort_geom
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  Node** transparent_node_list, **transparent_leaf_list;
  checkCudaErrors(hipMallocManaged(
    (void **)&transparent_node_list, 
    max(1, (num_transparent_geom[0] - 1)) * sizeof(Node *)));
  checkCudaErrors(hipMallocManaged(
    (void **)&transparent_leaf_list, 
    max(1, num_transparent_geom[0]) * sizeof(Node *)));

  start = clock();
  process = "Building transparent leaves";
  print_start_process(process, start);
  build_leaf_list<<<max(1, num_transparent_geom[0]), 1>>>(
    transparent_leaf_list, transparent_geom_list, num_transparent_geom[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Building transparent nodes";
  print_start_process(process, start);
  build_node_list<<<max(1, num_transparent_geom[0]), 1>>>(
    transparent_node_list, num_transparent_geom[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  unsigned int *transparent_morton_code_list;
  checkCudaErrors(hipMallocManaged(
    (void **)&transparent_morton_code_list,
    max(1, num_transparent_geom[0]) * sizeof(unsigned int)));

  start = clock();
  process = "Extracting transparent morton codes";
  print_start_process(process, start);
  extract_morton_code_list<<<max(1, num_transparent_geom[0]), 1>>>(
    transparent_geom_list, transparent_morton_code_list, 
    num_transparent_geom[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Setting transparent node relationship";
  print_start_process(process, start);
  set_node_relationship<<<max(1, num_transparent_geom[0]), 1>>>(
    transparent_node_list, transparent_leaf_list, transparent_morton_code_list,
    num_transparent_geom[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipFree(transparent_morton_code_list));
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  start = clock();
  process = "Compute transparent node bounding boxes";
  print_start_process(process, start);
  compute_node_bounding_boxes<<<max(1, num_transparent_geom[0]), 1>>>(
    transparent_leaf_list, transparent_node_list, num_transparent_geom[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged((void **)&num_target_geom, sizeof(int)));

  start = clock();
  process = "Computing the number of target geometries";
  print_start_process(process, start);
  compute_num_target_geom<<<1, 1>>>(
    my_geom, num_triangles[0], num_target_geom
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged(
    (void **)&target_geom_list, num_target_geom[0] * sizeof(Primitive *)));

  start = clock();
  process = "Collecting target geometries";
  print_start_process(process, start);
  collect_target_geom<<<1, 1>>>(
    my_geom, num_triangles[0], target_geom_list
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged(
    (void **)&target_world_bounding_box, sizeof(BoundingBox *)));

  start = clock();
  process = "Computing the target world bounding box";
  print_start_process(process, start);
  compute_world_bounding_box<<<1, 1>>>(
    target_world_bounding_box, target_geom_list, num_target_geom[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Computing the morton code of every target geometry bounding box";
  print_start_process(process, start);
  compute_morton_code_batch<<<blocks_world, threads_world>>>(
    target_geom_list, target_world_bounding_box, num_target_geom[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Sorting the target geometries based on morton code";
  print_start_process(process, start);
  thrust::stable_sort(
    thrust::device, target_geom_list, target_geom_list + num_target_geom[0],
    sort_geom);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  Node** target_node_list, **target_leaf_list;
  checkCudaErrors(hipMallocManaged(
    (void **)&target_node_list, (num_target_geom[0] - 1) * sizeof(Node *)));
  checkCudaErrors(hipMallocManaged(
    (void **)&target_leaf_list, num_target_geom[0] * sizeof(Node *)));

  start = clock();
  process = "Building target leaves";
  print_start_process(process, start);
  build_leaf_list<<<max(1, num_target_geom[0]), 1>>>(
    target_leaf_list, target_geom_list, num_target_geom[0], true
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Building target nodes";
  print_start_process(process, start);
  build_node_list<<<max(1, num_target_geom[0]), 1>>>(
    target_node_list, num_target_geom[0], true
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  unsigned int *target_morton_code_list;
  checkCudaErrors(hipMallocManaged(
    (void **)&target_morton_code_list,
    max(1, num_target_geom[0]) * sizeof(unsigned int)));

  start = clock();
  process = "Extracting target morton codes";
  print_start_process(process, start);
  extract_morton_code_list<<<max(1, num_target_geom[0]), 1>>>(
    target_geom_list, target_morton_code_list, num_target_geom[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Setting target node relationship";
  print_start_process(process, start);
  set_node_relationship<<<max(1, num_target_geom[0]), 1>>>(
    target_node_list, target_leaf_list, target_morton_code_list,
    num_target_geom[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipFree(target_morton_code_list));
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  start = clock();
  process = "Compute target node bounding boxes";
  print_start_process(process, start);
  compute_node_bounding_boxes<<<max(1, num_target_geom[0]), 1>>>(
    target_leaf_list, target_node_list, num_target_geom[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  start = clock();
  process = "Compute target node bounding cones";
  print_start_process(process, start);
  compute_node_bounding_cones<<<max(1, num_target_geom[0]), 1>>>(
    target_leaf_list, target_node_list, num_target_geom[0]
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  checkCudaErrors(hipMallocManaged((void **)&image_output, image_size));

  checkCudaErrors(
    hipMallocManaged(
      (void **)&rand_state_image, num_pixels * sizeof(hiprandState)));

  start = clock();
  process = "Generating hiprand state for rendering";
  print_start_process(process, start);
  init_curand_state<<<num_pixels / 8 + 1, 8>>>(num_pixels, rand_state_image);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  print_end_process(process, start);

  if (render_mode == 1) {
  
    start = clock();
    process = "Doing first pass for SSS objects";
    print_start_process(process, start);
    do_sss_first_pass<<<max(1, num_sss_points), 1>>>(
      sss_pts, num_sss_points,
      pathtracing_sample_size,
      pathtracing_level, sky_emission,
      bg_height, bg_width,
      bg_texture_r, bg_texture_g, bg_texture_b, node_list,
      rand_state_sss, target_geom_list,
      target_node_list,
      target_leaf_list,
      num_target_geom[0],
      hittable_pdf_weight
    );
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    print_end_process(process, start);

    dim3 blocks(im_width / tx + 1, im_height / ty + 1);
    dim3 threads(tx, ty);

    start = clock();
    process = "Clearing image";
    print_start_process(process, start);
    clear_image<<<blocks, threads>>>(image_output, im_width, im_height);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    print_end_process(process, start);

    start = clock();
    process = "Creating point image";
    print_start_process(process, start);
    create_point_image<<<num_sss_points / tx + 1, tx>>>(
      image_output, my_camera, sss_pts, num_sss_points
    );
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    print_end_process(process, start);

    start = clock();
    process = "Saving pts image";
    print_start_process(process, start);
    save_image(
      image_output, im_width, im_height, image_output_path + "_pts.ppm");
    print_end_process(process, start);
    checkCudaErrors(hipDeviceSynchronize());

    start = clock();
    process = "Rendering";
    print_start_process(process, start);
    path_tracing_render<<<blocks, threads>>>(
      image_output, my_camera, rand_state_image, pathtracing_sample_size,
      pathtracing_level, dof_sample_size,
      sky_emission, bg_height, bg_width,
      bg_texture_r, bg_texture_g, bg_texture_b, node_list, my_objects,
      sss_pts_node_list,
      target_node_list,
      target_leaf_list,
      target_geom_list, 
      num_target_geom[0],
      hittable_pdf_weight
    );
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    print_end_process(process, start);
  
  } else if (render_mode == 2) {
    PPMHitPoint **hit_point_list;
    Point **photon_list, **surface_photon_list, **volume_photon_list;
    vec3 *image_dir, *image_indir, *image_surface_photon, *image_volume_photon;

    checkCudaErrors(hipMallocManaged((void **)&image_dir, image_size));
    checkCudaErrors(hipMallocManaged((void **)&image_indir, image_size));
    checkCudaErrors(hipMallocManaged((void **)&image_surface_photon, image_size));
    checkCudaErrors(hipMallocManaged((void **)&image_volume_photon, image_size));

    checkCudaErrors(
      hipMallocManaged((void **)&hit_point_list, 
      num_pixels * sizeof(PPMHitPoint*)));

    checkCudaErrors(
      hipMallocManaged(
        (void **)&photon_list, ppm_num_photon_per_pass * sizeof(Point*)
      )
    );
    checkCudaErrors(
      hipMallocManaged(
        (void **)&surface_photon_list, ppm_num_photon_per_pass * sizeof(Point*)
      )
    );
    checkCudaErrors(
      hipMallocManaged(
        (void **)&volume_photon_list, ppm_num_photon_per_pass * sizeof(Point*)
      )
    );

    dim3 blocks(im_width / tx + 1, im_height / ty + 1);
    dim3 threads(tx, ty);

    start = clock();
    process = "Ray tracing pass";
    print_start_process(process, start);
    ray_tracing_pass<<<blocks, threads>>>(
      hit_point_list, my_camera, rand_state_image, nullptr, node_list, true, 
      ppm_max_bounce, ppm_alpha, 0,
      num_target_geom[0],
      target_geom_list,
      target_node_list,
      target_leaf_list,
      transparent_node_list,
      pathtracing_sample_size,
      ppm_radius_scaling_factor
    );
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    print_end_process(process, start);

    float *average_hit_point_radius;
    checkCudaErrors(
      hipMallocManaged(
        (void **)&average_hit_point_radius, sizeof(float)
      )
    );

    start = clock();
    process = "Compute average hit point radius";
    print_start_process(process, start);
    compute_average_radius<<<1, 1>>>(
      hit_point_list, num_pixels, average_hit_point_radius
    );
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    print_end_process(process, start);

    start = clock();
    process = "Assign radius to invalid hit points";
    print_start_process(process, start);
    assign_radius_to_invalid_hit_points<<<num_pixels, 1>>>(
      hit_point_list, num_pixels, average_hit_point_radius[0]);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    print_end_process(process, start);

    start = clock();
    process = "Clearing image";
    print_start_process(process, start);
    clear_image<<<blocks, threads>>>(image_output, im_width, im_height);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    print_end_process(process, start);

    start = clock();
    process = "Creating hit point image";
    print_start_process(process, start);
    create_point_image<<<num_pixels / tx + 1, tx>>>(
      image_output, my_camera, hit_point_list, num_pixels
    );
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    print_end_process(process, start);

    start = clock();
    process = "Saving hit point image";
    print_start_process(process, start);
    save_image(
      image_output, im_width, im_height, image_output_path + "_hit_point.ppm");
    print_end_process(process, start);
    checkCudaErrors(hipDeviceSynchronize());

    start = clock();
    process = "Create photon list";
    print_start_process(process, start);
    create_photon_list<<<ppm_num_photon_per_pass, 1>>>(
      photon_list, ppm_num_photon_per_pass);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    print_end_process(process, start);

    Node **surface_photon_node_list, **surface_photon_leaf_list;
    Node **volume_photon_node_list, **volume_photon_leaf_list;

    checkCudaErrors(hipMallocManaged(
      (void **)&surface_photon_node_list, 
      (ppm_num_photon_per_pass - 1) * sizeof(Node *)));
    checkCudaErrors(hipMallocManaged(
      (void **)&surface_photon_leaf_list,
      ppm_num_photon_per_pass * sizeof(Node *)));
    checkCudaErrors(hipMallocManaged(
      (void **)&volume_photon_node_list, 
      (ppm_num_photon_per_pass - 1) * sizeof(Node *)));
    checkCudaErrors(hipMallocManaged(
      (void **)&volume_photon_leaf_list,
      ppm_num_photon_per_pass * sizeof(Node *)));

    float *accummulated_target_geom_energy;
    checkCudaErrors(hipMallocManaged(
      (void **)&accummulated_target_geom_energy, 
      num_target_geom[0] * sizeof(float)));

    start = clock();
    process = "Init surface photon leaves";
    print_start_process(process, start);
    init_photon_leaves<<<ppm_num_photon_per_pass, 1>>>(
      surface_photon_leaf_list, ppm_num_photon_per_pass);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    print_end_process(process, start);

    start = clock();
    process = "Init surface photon nodes";
    print_start_process(process, start);
    init_photon_nodes<<<ppm_num_photon_per_pass, 1>>>(
      surface_photon_node_list, ppm_num_photon_per_pass);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    print_end_process(process, start);

    start = clock();
    process = "Init volume photon leaves";
    print_start_process(process, start);
    init_photon_leaves<<<ppm_num_photon_per_pass, 1>>>(
      volume_photon_leaf_list, ppm_num_photon_per_pass);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    print_end_process(process, start);

    start = clock();
    process = "Init volume photon nodes";
    print_start_process(process, start);
    init_photon_nodes<<<ppm_num_photon_per_pass, 1>>>(
      volume_photon_node_list, ppm_num_photon_per_pass);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    print_end_process(process, start);

    start = clock();
    process = "Compute accummulated light source energy";
    print_start_process(process, start);
    compute_accummulated_light_source_energy<<<1, 1>>>(
      target_geom_list, num_target_geom[0], accummulated_target_geom_energy);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    print_end_process(process, start);

    hiprandState *rand_state_ppm;
    size_t rand_state_ppm_size = ppm_num_photon_per_pass * sizeof(hiprandState);
    checkCudaErrors(hipMallocManaged(
      (void **)&rand_state_ppm, rand_state_ppm_size));

    int *num_surface_photons, *num_volume_photons;
    checkCudaErrors(hipMallocManaged(
      (void **)&num_surface_photons, sizeof(int)));
    checkCudaErrors(hipMallocManaged(
      (void **)&num_volume_photons, sizeof(int)));

    start = clock();
    process = "Generating hiprand state for photon shooting";
    print_start_process(process, start);
    init_curand_state<<<ppm_num_photon_per_pass, 1>>>(
      ppm_num_photon_per_pass, rand_state_ppm);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    print_end_process(process, start);

    unsigned int *surface_photon_morton_code_list;
    unsigned int *volume_photon_morton_code_list;
    checkCudaErrors(hipMallocManaged(
      (void **)&surface_photon_morton_code_list,
      max(1, ppm_num_photon_per_pass) * sizeof(unsigned int)));
    checkCudaErrors(hipMallocManaged(
      (void **)&volume_photon_morton_code_list,
      max(1, ppm_num_photon_per_pass) * sizeof(unsigned int)));

    for (int i = 0; i < ppm_num_pass; i++) {

      printf("PPM Pass %d.\n", i);

      start = clock();
      process = "Photon pass";
      print_start_process(process, start);
      photon_pass<<<ppm_num_photon_per_pass, 1>>>(
        target_geom_list, node_list, transparent_node_list, photon_list, 
	num_target_geom[0], accummulated_target_geom_energy, 
	ppm_num_photon_per_pass, ppm_max_bounce, i, rand_state_ppm);
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipDeviceSynchronize());
      print_end_process(process, start);

      start = clock();
      process = "Gather recorded photon";
      print_start_process(process, start);
      gather_recorded_photons<<<1, 1>>>(
        photon_list, surface_photon_list, volume_photon_list,
	ppm_num_photon_per_pass, num_surface_photons, num_volume_photons);
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipDeviceSynchronize());
      print_end_process(process, start);

      start = clock();
      process = "Clearing image";
      print_start_process(process, start);
      clear_image<<<blocks, threads>>>(image_surface_photon, im_width, im_height);
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipDeviceSynchronize());
      print_end_process(process, start);

      start = clock();
      process = "Clearing image";
      print_start_process(process, start);
      clear_image<<<blocks, threads>>>(
	image_volume_photon, im_width, im_height);
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipDeviceSynchronize());
      print_end_process(process, start);
      start = clock();

      process = "Creating photon image";
      print_start_process(process, start);
      create_point_image<<<num_surface_photons[0] / tx + 1, tx>>>(
        image_surface_photon, my_camera, surface_photon_list, 
	num_surface_photons[0]
      );
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipDeviceSynchronize());
      print_end_process(process, start);

      start = clock();
      process = "Creating photon image";
      print_start_process(process, start);
      create_point_image<<<num_volume_photons[0] / tx + 1, tx>>>(
        image_volume_photon, my_camera, volume_photon_list, 
	num_volume_photons[0]
      );
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipDeviceSynchronize());
      print_end_process(process, start);

      if (num_surface_photons[0] > 0) {
      	checkCudaErrors(hipDeviceSynchronize());
      	start = clock();
      	process = "Computing photon morton codes";
      	print_start_process(process, start);
      	compute_photon_morton_code_batch<<<num_surface_photons[0], 1>>>(
      	  surface_photon_list, num_surface_photons[0], world_bounding_box);
      	checkCudaErrors(hipGetLastError());
      	checkCudaErrors(hipDeviceSynchronize());
      	print_end_process(process, start);

      	process = "Sorting the photons based on morton code";
      	start = clock();
      	print_start_process(process, start);
      	thrust::stable_sort(
      	  thrust::device, surface_photon_list, 
      	  surface_photon_list + num_surface_photons[0],
      	  sort_points);
      	checkCudaErrors(hipGetLastError());
      	checkCudaErrors(hipDeviceSynchronize());
      	print_end_process(process, start);

      	start = clock();
      	process = "Reset photon nodes";
      	print_start_process(process, start);
      	reset_photon_nodes<<<ppm_num_photon_per_pass, 1>>>(
      	  surface_photon_node_list, ppm_num_photon_per_pass);
      	checkCudaErrors(hipGetLastError());
      	checkCudaErrors(hipDeviceSynchronize());
      	print_end_process(process, start);

      	process = "Assign photons to leaves";
      	start = clock();
      	print_start_process(process, start);
      	assign_photons<<<max(1, num_surface_photons[0]), 1>>>(
      	  surface_photon_leaf_list, surface_photon_list, 
      	  num_surface_photons[0]);
      	checkCudaErrors(hipGetLastError());
      	checkCudaErrors(hipDeviceSynchronize());
      	print_end_process(process, start);

      	start = clock();
      	process = "Extracting the morton codes of the photons";
      	print_start_process(process, start);
      	extract_sss_morton_code_list<<<max(1, num_surface_photons[0]), 1>>>(
      	  surface_photon_list, surface_photon_morton_code_list, 
      	  num_surface_photons[0]
      	);
      	checkCudaErrors(hipGetLastError());
      	checkCudaErrors(hipDeviceSynchronize());
      	print_end_process(process, start);

      	process = "Setting photon node relationship";
      	start = clock();
      	print_start_process(process, start);
      	set_photon_node_relationship<<<max(1, num_surface_photons[0]), 1>>>(
      	  surface_photon_node_list, surface_photon_leaf_list, 
      	  surface_photon_morton_code_list, num_surface_photons[0]
      	);
      	checkCudaErrors(hipGetLastError());
      	checkCudaErrors(hipDeviceSynchronize());
      	print_end_process(process, start);

      	start = clock();
      	process = "Compute node bounding boxes";
      	print_start_process(process, start);
      	compute_node_bounding_boxes<<<max(1, num_surface_photons[0]), 1>>>(
      	  surface_photon_leaf_list, surface_photon_node_list, 
      	  num_surface_photons[0]
      	);
      	checkCudaErrors(hipGetLastError());
      	checkCudaErrors(hipDeviceSynchronize());
      	print_end_process(process, start);
      }

      if (num_volume_photons[0] > 0) {
      	checkCudaErrors(hipDeviceSynchronize());
      	start = clock();
      	process = "Computing photon morton codes";
      	print_start_process(process, start);
      	compute_photon_morton_code_batch<<<num_volume_photons[0], 1>>>(
      	  volume_photon_list, num_volume_photons[0], world_bounding_box);
      	checkCudaErrors(hipGetLastError());
      	checkCudaErrors(hipDeviceSynchronize());
      	print_end_process(process, start);

      	process = "Sorting the photons based on morton code";
      	start = clock();
      	print_start_process(process, start);
      	thrust::stable_sort(
      	  thrust::device, volume_photon_list, 
      	  volume_photon_list + num_volume_photons[0],
      	  sort_points);
      	checkCudaErrors(hipGetLastError());
      	checkCudaErrors(hipDeviceSynchronize());
      	print_end_process(process, start);

      	start = clock();
      	process = "Reset photon nodes";
      	print_start_process(process, start);
      	reset_photon_nodes<<<ppm_num_photon_per_pass, 1>>>(
      	  volume_photon_node_list, ppm_num_photon_per_pass);
      	checkCudaErrors(hipGetLastError());
      	checkCudaErrors(hipDeviceSynchronize());
      	print_end_process(process, start);

      	process = "Assign photons to leaves";
      	start = clock();
      	print_start_process(process, start);
      	assign_photons<<<max(1, num_volume_photons[0]), 1>>>(
      	  volume_photon_leaf_list, volume_photon_list, 
      	  num_volume_photons[0]);
      	checkCudaErrors(hipGetLastError());
      	checkCudaErrors(hipDeviceSynchronize());
      	print_end_process(process, start);

      	start = clock();
      	process = "Extracting the morton codes of the photons";
      	print_start_process(process, start);
      	extract_sss_morton_code_list<<<max(1, num_volume_photons[0]), 1>>>(
      	  volume_photon_list, volume_photon_morton_code_list, 
      	  num_volume_photons[0]
      	);
      	checkCudaErrors(hipGetLastError());
      	checkCudaErrors(hipDeviceSynchronize());
      	print_end_process(process, start);

      	process = "Setting photon node relationship";
      	start = clock();
      	print_start_process(process, start);
      	set_photon_node_relationship<<<max(1, num_volume_photons[0]), 1>>>(
      	  volume_photon_node_list, volume_photon_leaf_list, 
      	  volume_photon_morton_code_list, num_volume_photons[0]
      	);
      	checkCudaErrors(hipGetLastError());
      	checkCudaErrors(hipDeviceSynchronize());
      	print_end_process(process, start);

      	start = clock();
      	process = "Compute node bounding spheres";
      	print_start_process(process, start);
      	compute_node_bounding_spheres<<<max(1, num_volume_photons[0]), 1>>>(
      	  volume_photon_leaf_list, volume_photon_node_list, 
      	  num_volume_photons[0]
      	);
      	checkCudaErrors(hipGetLastError());
      	checkCudaErrors(hipDeviceSynchronize());
      	print_end_process(process, start);
      }

      printf("Number of surface photons = %d\n", num_surface_photons[0]);
      printf("Number of volume photons  = %d\n", num_volume_photons[0]);

      start = clock();
      process = "Ray tracing pass";
      print_start_process(process, start);
      ray_tracing_pass<<<blocks, threads>>>(
        hit_point_list, my_camera, rand_state_image, volume_photon_node_list, 
	node_list, false, 
        ppm_max_bounce, ppm_alpha, i,
        num_target_geom[0],
        target_geom_list,
	target_node_list,
        target_leaf_list,
	transparent_node_list,
        pathtracing_sample_size,
	ppm_radius_scaling_factor
      );
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipDeviceSynchronize());
      print_end_process(process, start);

      process = "Update hit point parameters";
      start = clock();
      print_start_process(process, start);
      update_hit_point_parameters<<<num_pixels, 1>>>(
        i + 1, surface_photon_node_list, node_list, hit_point_list, num_pixels, 
	ppm_num_photon_per_pass
      );
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipDeviceSynchronize());
      print_end_process(process, start);

      start = clock();
      process = "Compute average hit point radius";
      print_start_process(process, start);
      compute_average_radius<<<1, 1>>>(
        hit_point_list, num_pixels, average_hit_point_radius
      );
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipDeviceSynchronize());
      print_end_process(process, start);

      printf("The average hit point radius is %f.\n", average_hit_point_radius[0]);

      process = "Compute direct lighting image output";
      start = clock();
      print_start_process(process, start);
      get_ppm_image_output<<<blocks, threads>>>(
        i + 1, image_dir, hit_point_list, my_camera, 0
      );
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipDeviceSynchronize());
      print_end_process(process, start);

      process = "Compute indirect lighting image output";
      start = clock();
      print_start_process(process, start);
      get_ppm_image_output<<<blocks, threads>>>(
        i + 1, image_indir, hit_point_list, my_camera, 1
      );
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipDeviceSynchronize());
      print_end_process(process, start);

      process = "Compute image output";
      start = clock();
      print_start_process(process, start);
      get_ppm_image_output<<<blocks, threads>>>(
        i + 1, image_output, hit_point_list, my_camera, 2
      );
      checkCudaErrors(hipGetLastError());
      checkCudaErrors(hipDeviceSynchronize());
      print_end_process(process, start);

      if (i % ppm_image_output_iteration == 0) {
          start = clock();
          process = "Saving photon image";
          print_start_process(process, start);
          save_image(
            image_surface_photon, im_width, im_height, 
            image_output_path + "_surface_photon.ppm");
          print_end_process(process, start);

          start = clock();
          process = "Saving photon image";
          print_start_process(process, start);
          save_image(
            image_volume_photon, im_width, im_height, 
            image_output_path + "_volume_photon.ppm");
          print_end_process(process, start);

          start = clock();
          process = "Saving direct radiance image";
          print_start_process(process, start);
          save_image(image_dir, im_width, im_height, image_output_path + "_direct.ppm");
          print_end_process(process, start);
          checkCudaErrors(hipDeviceSynchronize());

          start = clock();
          process = "Saving indirect radiance image";
          print_start_process(process, start);
          save_image(image_indir, im_width, im_height, image_output_path + "_indirect.ppm");
          print_end_process(process, start);
          checkCudaErrors(hipDeviceSynchronize());

          start = clock();
          process = "Saving image";
          print_start_process(process, start);
          save_image(image_output, im_width, im_height, image_output_path + "_global.ppm");
          print_end_process(process, start);
          checkCudaErrors(hipDeviceSynchronize());
      }

    }
  } 

  start = clock();
  process = "Saving image";
  print_start_process(process, start);
  save_image(image_output, im_width, im_height, image_output_path + ".ppm");
  print_end_process(process, start);
  checkCudaErrors(hipDeviceSynchronize());

  start = clock();
  process = "Cleaning";
  print_start_process(process, start);
  // free_world<<<1,1>>>(my_scene, my_grid, my_geom, my_camera, max_num_faces);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipFree(my_camera));
  checkCudaErrors(hipFree(my_geom));
  checkCudaErrors(hipFree(my_material));
  checkCudaErrors(hipFree(num_triangles));
  checkCudaErrors(hipFree(rand_state_image));
  checkCudaErrors(hipFree(rand_state_sss));
  checkCudaErrors(hipFree(material_image_r));
  checkCudaErrors(hipFree(material_image_g));
  checkCudaErrors(hipFree(material_image_b));
  checkCudaErrors(hipFree(bg_texture_r));
  checkCudaErrors(hipFree(bg_texture_g));
  checkCudaErrors(hipFree(bg_texture_b));
  checkCudaErrors(hipFree(image_output));
  print_end_process(process, start);

  print_end_process("Rendering project", first_start);

  hipDeviceReset();

  return 0;
}
