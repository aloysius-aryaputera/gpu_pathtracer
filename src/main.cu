#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <string>

#include "model/camera.h"
#include "model/data_structure/local_vector.h"
#include "model/geometry/sphere.h"
#include "model/geometry/triangle.h"
#include "model/grid/bounding_box.h"
#include "model/grid/cell.h"
#include "model/grid/grid.h"
#include "model/material.h"
#include "model/ray.h"
#include "model/scene.h"
#include "model/vector_and_matrix/vec3.h"
#include "render/pathtracing.h"
#include "util/image_util.h"
#include "util/read_file_util.h"
#include "world_lib.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void create_scene(
  Scene** scene, Camera** camera, Grid** grid, int *num_objects
) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    *(scene) = new Scene(camera[0], grid[0], num_objects[0]);
  }
}

__global__ void render_init(
  int im_width, int im_height, hiprandState *rand_state, int *progress
) {
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  if (i == 0 && j == 0) {
    progress[0] = 0;
  }
  if ((j >= im_width) || (i >= im_height)) {
    return;
  }
  int pixel_index = i * im_width + j;
  //Each thread gets same seed, a different sequence number, no offset
  hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void free_world(
  Scene** scene, Grid **grid, Primitive **geom_array, Camera **camera, int n
) {
    for (int i = 0; i < n; i++){
      delete *(geom_array + i);
    }
    delete *camera;
    delete *grid;
    delete *scene;
}

int main(int argc, char **argv) {
  int im_width = std::stoi(argv[3]), im_height = std::stoi(argv[4]);
  int tx = std::stoi(argv[5]), ty = std::stoi(argv[6]);
  int *n_cell_x, *n_cell_y, *n_cell_z;
  int max_n_cell_x = 60, max_n_cell_y = 60, max_n_cell_z = 60;
  int tx2 = 32, ty2 = 32, max_num_objects_per_cell = 500, *progress;

  printf("im_width = %d, im_height = %d\n", im_width, im_height);
  printf("tx = %d, ty = %d\n", tx, ty);

  Scene** my_scene;
  Grid** my_grid;
  Cell** my_cell;
  Primitive **my_geom, **my_cell_geom;
  Camera **my_camera;
  vec3 *fb;
  int num_pixels = im_width * im_height;
  int max_num_vertices = 60000, max_num_faces = 110000;
  size_t fb_size = num_pixels * sizeof(vec3);
  hiprandState *rand_state;
  size_t rand_state_size = num_pixels * sizeof(hiprandState);
  size_t cell_geom_size = max_num_objects_per_cell * (max_n_cell_x) * \
    (max_n_cell_y) * (max_n_cell_z) * sizeof(Primitive*);
  clock_t start, stop;

  start = clock();

  float *x, *y, *z;
  int *point_1_idx, *point_2_idx, *point_3_idx;
  int *num_triangles;

  checkCudaErrors(hipMallocManaged((void **)&num_triangles, sizeof(int)));

  checkCudaErrors(hipMallocManaged((void **)&x, max_num_vertices * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&y, max_num_vertices * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&z, max_num_vertices * sizeof(float)));

  checkCudaErrors(hipMallocManaged((void **)&point_1_idx, max_num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&point_2_idx, max_num_faces * sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&point_3_idx, max_num_faces * sizeof(int)));

  extract_triangle_data(
    argv[2], x, y, z, point_1_idx, point_2_idx, point_3_idx, num_triangles
  );

  checkCudaErrors(hipMallocManaged((void **)&my_geom, max_num_faces * sizeof(Primitive *)));
  checkCudaErrors(hipMallocManaged((void **)&my_camera, sizeof(Camera *)));

  printf("Creating the world!\n");
  create_world_2<<<1, 1>>>(
    my_camera, my_geom, x, y, z, point_1_idx, point_2_idx, point_3_idx,
    num_triangles, im_width, im_height
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  printf("World created!\n");
  checkCudaErrors(hipFree(x));
  checkCudaErrors(hipFree(y));
  checkCudaErrors(hipFree(z));
  checkCudaErrors(hipFree(point_1_idx));
  checkCudaErrors(hipFree(point_2_idx));
  checkCudaErrors(hipFree(point_3_idx));
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMallocManaged((void **)&my_grid, sizeof(Grid *)));
  checkCudaErrors(hipMallocManaged((void **)&my_cell, max_n_cell_x * max_n_cell_y * max_n_cell_z * sizeof(Cell *)));
  checkCudaErrors(hipMallocManaged((void **)&n_cell_x, sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&n_cell_y, sizeof(int)));
  checkCudaErrors(hipMallocManaged((void **)&n_cell_z, sizeof(int)));
  printf("Creating the grid!\n");
  create_grid<<<1, 1>>>(
    my_camera, my_grid, my_geom, num_triangles, my_cell, n_cell_x, n_cell_y,
    n_cell_z, max_n_cell_x, max_n_cell_y, max_n_cell_z, max_num_objects_per_cell
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());
  printf("Grid created!\n");

  dim3 blocks2(n_cell_x[0] / tx2 + 1, n_cell_y[0] / ty2 + 1);
  dim3 threads2(tx2, ty2);
  checkCudaErrors(hipMallocManaged((void **)&my_cell_geom, cell_geom_size));
  build_cell_array<<<blocks2, threads2>>>(my_grid, my_cell_geom);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  insert_objects<<<blocks2, threads2>>>(my_grid);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  checkCudaErrors(hipMallocManaged((void **)&my_scene, sizeof(Scene *)));
  create_scene<<<1, 1>>>(my_scene, my_camera, my_grid, num_triangles);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  dim3 blocks(im_width / tx + 1, im_height / ty + 1);
  dim3 threads(tx, ty);
  checkCudaErrors(hipMallocManaged((void **)&rand_state, rand_state_size));
  checkCudaErrors(hipMallocManaged((void **)&progress, sizeof(int)));
  render_init<<<blocks, threads>>>(im_width, im_height, rand_state, progress);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  vec3 sky_emission = vec3(1, 1, 1);
  checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));
  render<<<blocks, threads>>>(
    fb, my_scene, rand_state, std::stoi(argv[7]), std::stoi(argv[8]),
    sky_emission, progress
  );
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  printf("Saving image!\n");
  save_image(fb, im_width, im_height, argv[1]);
  printf("Image saved!\n");

  stop = clock();
  double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
  printf("\nThe rendering took %5.5f seconds.\n", timer_seconds);

  checkCudaErrors(hipDeviceSynchronize());
  free_world<<<1,1>>>(my_scene, my_grid, my_geom, my_camera, 9999);
  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipFree(my_scene));
  checkCudaErrors(hipFree(my_grid));
  checkCudaErrors(hipFree(my_camera));
  checkCudaErrors(hipFree(my_geom));
  checkCudaErrors(hipFree(num_triangles));
  checkCudaErrors(hipFree(n_cell_x));
  checkCudaErrors(hipFree(n_cell_y));
  checkCudaErrors(hipFree(n_cell_z));
  checkCudaErrors(hipFree(rand_state));
  checkCudaErrors(hipFree(fb));

  hipDeviceReset();

  return 0;
}
